#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>

hipError_t DistanceMatrixWithCuda(char* T, char* P, uint16_t* dMatrix, const int tSize, const int pSize);

__host__ void WrongArgsPrint();

__global__ void CalculateDistanceMatrixKernel(char* T, char* P, uint16_t* dMatrix, int pSize, int tSize)
{
	uint16_t global_tid = threadIdx.x + blockDim.x * blockIdx.x;
	uint8_t lane_id = threadIdx.x % warpSize;
	uint16_t tid = threadIdx.x;
	for (uint16_t row = 0; row <= pSize; row++) {

	}
}

int main(int argc, char** argv)
{
	if (argc != 3) {
		WrongArgsPrint();
		return 1;
	}
	
	hipError_t cudaStatus = DistanceMatrixWithCuda();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "addWithCuda failed!");
		return 1;
	}

	cudaStatus = hipDeviceReset();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceReset failed!");
		return 1;
	}

	return 0;
}

// Helper function for using CUDA to add vectors in parallel.
hipError_t DistanceMatrixWithCuda(char* T, char* P, uint16_t* dMatrix, const int tSize, const int pSize)
{
	char* dev_T;
	char* dev_P;
	uint16_t* dev_dMatrix;
	hipError_t cudaStatus;

	// Choose which GPU to run on, change this on a multi-GPU system.
	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		goto Error;
	}

	// Allocate GPU buffers for three vectors (two input, one output)    .
	cudaStatus = hipMalloc((void**)&dev_T, tSize * sizeof(char));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&dev_P, pSize * sizeof(char));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}
	cudaStatus = hipMalloc((void**)&dev_dMatrix, (pSize + 1) * (tSize + 1) * sizeof(uint16_t));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}
	// Copy input vectors from host memory to GPU buffers.
	cudaStatus = hipMemcpy(dev_T, T, tSize * sizeof(char), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	cudaStatus = hipMemcpy(dev_P, P, pSize * sizeof(char), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	// Launch a kernel on the GPU with one thread for each element.
	int threadsPerBlock = 1024;
	int totalThreads = tSize; 
	int blocks = (totalThreads + threadsPerBlock - 1) / threadsPerBlock;
	CalculateDistanceMatrixKernel << <blocks, threadsPerBlock >> > (dev_T, dev_P, dev_dMatrix);

	// Check for any errors launching the kernel
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}

	// hipDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
		goto Error;
	}

	// Copy output vector from GPU buffer to host memory.
	cudaStatus = hipMemcpy(dMatrix, dev_dMatrix, (pSize + 1) * (tSize + 1) * sizeof(uint16_t), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

Error:
	hipFree(dev_P);
	hipFree(dev_T);
	hipFree(dev_dMatrix);

	return cudaStatus;
}

__host__ void WrongArgsPrint() {
	printf("Correct way to invoke program is: \"filename s1 s2\"");
}

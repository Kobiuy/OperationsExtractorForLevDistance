#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include ""

// compute-sanitizer OperationsExtractorForLevDistance.exe catgactg tactg> a.txt 2>&1

// TODO String Form
#include <stdio.h>
#include <cmath>
#include <string>
#include <cstring>
#include <fstream>
#include <iomanip>
#include <iostream>
#include <stack>
#include <chrono>
#define A_SIZE 26
using namespace std;
using namespace std::chrono;

__host__ hipError_t DistanceMatrixWithCuda(const char* T, const char* P, int* dMatrix, int* xMatrix, const size_t tSize, const size_t pSize);
__host__ int main(int argc, char** argv);
__host__ void ReadFile(const char* filename, string* line1, string* line2);
__host__ hipError_t XMatrixWithCuda(const char* T, int* xMatrix, const size_t tSize);
__host__ string CalculatePathFromD(int* dMatrix, const char* T, const char* P, const size_t tSize, const size_t pSize, int* distance);
__host__ void WriteToFile(string result, int distance);
__host__ void WrongArgsPrint();
__host__ void PrintMatrix(int* matrix, size_t height, size_t width);
__host__ void PrintMatrixToFile(int* matrix, size_t height, size_t width);

__global__ void CalculateXMatrixKernel(char* T, int* xMatrix, size_t* tSize) {
	int global_tid = threadIdx.x + blockDim.x * blockIdx.x;
	uint8_t aSize = 26;
	int firstInRow = global_tid * (*tSize + 1);
	xMatrix[firstInRow] = 0;
	for (int i = 1; i <= *tSize; ++i) {
		;		if (T[i] == global_tid + 'A') {
			xMatrix[i + firstInRow] = i;
		}
		else {
			xMatrix[i + firstInRow] = xMatrix[i + firstInRow - 1];
		}
	}
}

__global__ void CalculateDistanceMatrixKernel(char* T, char* P, int* xMatrix, int* dMatrix, size_t* pSize, size_t* tSize)
{
	size_t pSizeLocal = *pSize;
	size_t tSizeLocal = *tSize;
	int global_tid = threadIdx.x + blockDim.x * blockIdx.x;
	int tid = threadIdx.x;
	uint8_t lane_id = threadIdx.x % 32;
	int warpId = threadIdx.x / 32;

	extern __shared__ char p[];
	char t = T[global_tid];
	int Dvar = 0;
	int Bvar = 0;
	int Cvar = 0;
	int Avar = 0;
	int Xvar = 0;

	for (int i = tid; i <= pSizeLocal; i += blockDim.x) {
		p[i] = P[i];
	}
	if (global_tid > tSizeLocal) return;

	for (int row = 0; row <= pSizeLocal; ++row) {

		__syncthreads();
		Avar = __shfl_up(Dvar, 1);
		if (lane_id == 0 && global_tid != 0) { // waiting for other blocks
			while (dMatrix[(row - 1) * (tSizeLocal + 1) + (global_tid - 1)] == -1) {}
			Avar = dMatrix[(row - 1) * (tSizeLocal + 1) + (global_tid - 1)];
		}

		Bvar = Dvar;

		if (row == 0) {
			Dvar = global_tid;
		}
		else if (global_tid == 0) {
			Dvar = row;
		}
		else if (t == p[row]) {
			Dvar = Avar;
		}
		else if (xMatrix[(p[row] - 'A') * (tSizeLocal + 1) + global_tid] == 0) {
			Dvar = 1 + min(Avar, min(Bvar, row + global_tid - 1));

		}
		else {
			Xvar = xMatrix[(p[row] - 'A') * (tSizeLocal + 1) + global_tid];
			Cvar = dMatrix[(row - 1) * (tSizeLocal + 1) + (Xvar - 1)];
			Dvar = 1 + min(Avar, min(Bvar, Cvar + global_tid - 1 - Xvar));
		}

		dMatrix[row * (tSizeLocal + 1) + global_tid] = Dvar;
	}
}

int main(int argc, char** argv)
{
	if (argc > 2) {
		WrongArgsPrint();
		return 1;
	}
	const char* filename = "dane.txt";
	if (argc == 2)
		filename = argv[1];


	// Read File
	string line1, line2;
	ReadFile(filename, &line1, &line2);
	if (line1 == "" || line2 == "") {
		cerr << "Error reading file" << endl;
		return 1;
	}

	const char* T = line1.c_str();
	const char* P = line2.c_str();

	const size_t tSize = line1.length();
	const size_t pSize = line2.length();
	int distance;

	int* dMatrix = (int*)malloc((tSize + 1) * (pSize + 1) * sizeof(int));
	if (dMatrix == NULL) {
		perror("Memory allocation failed");
		return 1;
	}
	for (size_t i = 0; i < pSize + 1; ++i) {
		for (size_t j = 0; j < tSize + 1; ++j) {
			dMatrix[i * (tSize + 1) + j] = -1;
		}
	}
	int* xMatrix = (int*)malloc((tSize + 1) * 26 * sizeof(int));

	hipError_t cudaStatus = XMatrixWithCuda(T, xMatrix, tSize);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "XMatrixWithCuda failed!");
		return 1;
	}
	//PrintMatrix(xMatrix, 26, tSize + 1);


	cudaStatus = DistanceMatrixWithCuda(T, P, dMatrix, xMatrix, tSize, pSize);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "DistanceMatrixWithCuda failed!");
		return 1;
	}
	cout << "D Calculated" << endl;

	//PrintMatrix(dMatrix, pSize + 1, tSize + 1);
	//PrintMatrixToFile(dMatrix, pSize + 1, tSize + 1);

	string result = CalculatePathFromD(dMatrix, T, P, tSize, pSize, &distance);
	std::cout << result << endl;
	std::cout << distance << endl;
	std::cout << "Distance: " << dMatrix[pSize * (tSize + 1) + tSize] << endl;

	WriteToFile(result, distance);

	cudaStatus = hipDeviceReset();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceReset failed!");
		return 1;
	}

	return 0;
}

__host__ hipError_t XMatrixWithCuda(const char* T, int* xMatrix, const size_t tSize)
{
	std::chrono::time_point<std::chrono::high_resolution_clock> ts;
	std::chrono::time_point<std::chrono::high_resolution_clock> te;

	char* dev_T;
	size_t* dev_tSize;
	int* dev_xMatrix;
	hipError_t cudaStatus;

	// Choose which GPU to run on, change this on a multi-GPU system.
	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		goto Error;
	}

	// Allocate GPU buffers for three vectors (two input, one output)    .
	cudaStatus = hipMalloc((void**)&dev_T, (tSize + 1) * sizeof(char));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}
	cudaStatus = hipMalloc((void**)&dev_xMatrix, A_SIZE * (tSize + 1) * sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}
	cudaStatus = hipMalloc((void**)&dev_tSize, sizeof(size_t));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}
	// Copy host to device
	cudaStatus = hipMemcpy(dev_T + 1, T, tSize * sizeof(char), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}
	cudaStatus = hipMemcpy(dev_tSize, &tSize, sizeof(size_t), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	// Launch a kernel on the GPU with one thread for each element.
	ts = high_resolution_clock::now();
	CalculateXMatrixKernel << <1, 26 >> > (dev_T, dev_xMatrix, dev_tSize);

	// Check for any errors launching the kernel
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}

	// hipDeviceSynchronize waits for the kernel to finish, and returns any errors encountered during the launch.
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
		goto Error;
	}

	te = high_resolution_clock::now();
	cout << "Time of CalculateDistanceMatrixKernel:    " << setw(7) << 0.001 * duration_cast<microseconds>(te - ts).count() << " nsec" << endl;

	// Copy output vector from GPU buffer to host memory.
	cudaStatus = hipMemcpy(xMatrix, dev_xMatrix, A_SIZE * (tSize + 1) * sizeof(int), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

Error:
	hipFree(dev_T);
	hipFree(dev_xMatrix);
	hipFree(dev_tSize);

	return cudaStatus;
}

__host__ hipError_t DistanceMatrixWithCuda(const char* T, const char* P, int* dMatrix, int* xMatrix, const size_t tSize, const size_t pSize)
{
	std::chrono::time_point<std::chrono::high_resolution_clock> ts;
	std::chrono::time_point<std::chrono::high_resolution_clock> te;

	char* dev_T;
	char* dev_P;
	int* dev_dMatrix;
	size_t* dev_tSize;
	size_t* dev_pSize;
	hipError_t cudaStatus;
	int* dev_xMatrix;
	int threadsPerBlock = 1024;
	size_t totalThreads = tSize + 1;
	int blocks = (totalThreads + threadsPerBlock - 1) / threadsPerBlock;
	cout << "Blocks: " << blocks << " Threads: " << totalThreads << endl;
	// Choose which GPU to run on, change this on a multi-GPU system.
	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		goto Error;
	}

	// Allocate GPU buffers for three vectors (two input, one output)    .
	cudaStatus = hipMalloc((void**)&dev_T, (tSize + 1) * sizeof(char));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&dev_P, (pSize + 1) * sizeof(char));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&dev_dMatrix, (pSize + 1) * (tSize + 1) * sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}
	cudaStatus = hipMalloc((void**)&dev_xMatrix, A_SIZE * (tSize + 1) * sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}
	cudaStatus = hipMalloc((void**)&dev_tSize, sizeof(size_t));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}
	cudaStatus = hipMalloc((void**)&dev_pSize, sizeof(size_t));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}
	// Copy input vectors from host memory to GPU buffers.
	cudaStatus = hipMemcpy(dev_T + 1, T, tSize * sizeof(char), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	cudaStatus = hipMemcpy(dev_P + 1, P, pSize * sizeof(char), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}
	cudaStatus = hipMemcpy(dev_tSize, &tSize, sizeof(size_t), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}
	cudaStatus = hipMemcpy(dev_pSize, &pSize, sizeof(size_t), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}
	cudaStatus = hipMemcpy(dev_xMatrix, xMatrix, A_SIZE * (tSize + 1) * sizeof(int), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}
	cudaStatus = hipMemcpy(dev_dMatrix, dMatrix, (pSize + 1) * (tSize + 1) * sizeof(int), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	// Launch a kernel on the GPU with one thread for each element.
	ts = high_resolution_clock::now();
	CalculateDistanceMatrixKernel << <blocks, threadsPerBlock, pSize + 1 >> > (dev_T, dev_P, dev_xMatrix, dev_dMatrix, dev_pSize, dev_tSize);

	// Check for any errors launching the kernel
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}

	// hipDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
		goto Error;
	}
	te = high_resolution_clock::now();
	cout << "Time of CalculateDistanceMatrixKernel:    " << setw(7) << 0.001 * duration_cast<microseconds>(te - ts).count() << " nsec" << endl;

	// Copy output vector from GPU buffer to host memory.
	cudaStatus = hipMemcpy(dMatrix, dev_dMatrix, (pSize + 1) * (tSize + 1) * sizeof(int), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

Error:
	hipFree(dev_P);
	hipFree(dev_T);
	hipFree(dev_dMatrix);
	hipFree(dev_xMatrix);
	hipFree(dev_tSize);
	hipFree(dev_pSize);

	return cudaStatus;
}

__host__ string CalculatePathFromD(int* dMatrix, const char* T, const char* P, const size_t tSize, const size_t pSize, int* distance)
{
	size_t i = pSize;
	size_t j = tSize;
	string result;
	stack<string> path;

	*distance = 0;
	while (i != 0 || j != 0) {
		int minValue = 0;
		(*distance)++;
		if (i != 0 && j != 0)
			minValue = min(dMatrix[(i - 1) * (tSize + 1) + j], min(dMatrix[i * (tSize + 1) + (j - 1)], dMatrix[(i - 1) * (tSize + 1) + (j - 1)]));
		else if (j != 0)
			minValue = dMatrix[i * (tSize + 1) + (j - 1)];
		else
			minValue = dMatrix[(i - 1) * (tSize + 1) + j];
		if (i != 0 && j != 0 && minValue == dMatrix[(i - 1) * (tSize + 1) + (j - 1)]) {
			if (T[j - 1] == P[i - 1]) {
				//path.push("NO OPERATION\n");
				(*distance)--;
			}
			else {
				path.push(string("R, ") + to_string(j - 1) + ", " + P[i - 1] + "\n");
			}
			i--;
			j--;
		}
		else if (j != 0 && minValue == dMatrix[i * (tSize + 1) + (j - 1)]) {
			path.push(string("D, ") + to_string(j - 1) + ", " + T[j - 1] + "\n");
			j--;
		}
		else if (i != 0 && minValue == dMatrix[(i - 1) * (tSize + 1) + j]) {
			path.push(string("I, ") + to_string(j - 1) + ", " + P[i - 1] + "\n");
			i--;
		}
	}
	while (!path.empty()) {
		result.append(path.top());
		path.pop();
	}
	return result;
}
__host__ void WrongArgsPrint() {
	printf("Correct way to invoke program is: \"filename s1 s2\"");
}
__host__ void WriteToFile(string result, int distance) {
	ofstream myfile;
	myfile.open("result.txt");
	myfile << result << endl;
	myfile << "Distance: " << distance;
	myfile.close();
}
__host__ void ReadFile(const char* filename, string* line1, string* line2)
{
	ifstream file(filename);

	if (!file) {
		cerr << "Error opening file: " << filename << endl;
		return;
	}


	if (!getline(file, *line1))
		cerr << "Error reading first line or file is empty." << endl;
	if (!getline(file, *line2))
		cerr << "Error reading second line or file does not have a second line." << endl;
	file.close();
}
__host__ void PrintMatrix(int* matrix, size_t height, size_t width) {
	for (int j = 0; j < height; ++j) {
		for (int i = 0; i < width; ++i) {
			cout << setw(3) << matrix[j * width + i];
		}
		cout << endl;
	}
	cout << endl;
}
__host__ void PrintMatrixToFile(int* matrix, size_t height, size_t width) {
	ofstream myfile;
	myfile.open("DMatrix.txt");
	for (int j = 0; j < height; ++j) {
		for (int i = 0; i < width; ++i) {
			myfile << matrix[j * width + i];
		}
	}
	myfile.close();
}